#include "hip/hip_runtime.h"
#include "vars.h"
#include "cuda_runtime.hpp"

#include <hip/hip_runtime.h>
#define WARPSIZE 32
#define MINIBATCH 12
#define BLOCKSIZE 256
#define BUFFER 24

extern int neuron;
extern int layer;
extern int batch;
extern float bias;

extern int **csrdispl;
extern unsigned short **csrindex;
extern float **csrvalue;

extern float *currfeat;
extern float *nextfeat;
extern int *active;
extern int *categories;
extern int *globalcategories;

extern int myid;
extern int numproc;
extern int numthreads;

extern int *batchdispl;
extern int mybatch;
extern int extbatch;

int **buffdispl;
int **mapdispl;
int **warpdispl;
unsigned short **map;
unsigned short **warpindex;
float **warpvalue;

int **buffdispl_d;
int **mapdispl_d;
int **warpdispl_d;
unsigned short *mapbuff_d;
unsigned short *indbuff_d;
float *valbuff_d;;

#ifdef OUTOFCORE
int  weightsizemax;
int  mapsizemax;
#ifdef OVERLAP
unsigned short *mapstream_d;
unsigned short *indstream_d;
float *valstream_d;
#endif
#else
unsigned short **map_d;
unsigned short **warpindex_d;
float **warpvalue_d;
#endif

float *currfeat_d;
float *nextfeat_d;
int *active_d;
int *categories_d;

int numblocks;
int numwarp;
int buffsize;

hipStream_t copystream;
hipStream_t kernelstream;
float elapsedTime;

__device__ float __ReLU(float x) {
   return x<0.0?0.0:x>32.0?32.0:x;
};

__global__ void __launch_bounds__(1024,1) dummy_kernel(float *nextfeat, float *currfeat, int buffsize, int *buffdispl, int *mapdispl, unsigned short *map, int *displ, unsigned short *index, float *value, float bias , int neuron, int *categories, int *active) {
  extern __shared__ float shared[];
  int wind = threadIdx.x%WARPSIZE;
  float reduce[MINIBATCH] = {0.0};
  for (int buff = buffdispl[blockIdx.x]; buff < buffdispl[blockIdx.x+1]; buff++) {
    int mapnz = mapdispl[buff+1]-mapdispl[buff];
    for (int n = threadIdx.x; n < mapnz; n += blockDim.x) {
      int ind = map[mapdispl[buff]+n];
      for (unsigned int f = 0; f < MINIBATCH; f++)
        shared[f*buffsize+n] = currfeat[categories[blockIdx.y*MINIBATCH+f]* (unsigned int) neuron+ind];
    }
    __syncthreads();
    int warp = (buff*blockDim.x+threadIdx.x)/WARPSIZE;
    for (int m = displ[warp]; m < displ[warp+1]; m++) {
      int ind = index[m*WARPSIZE+wind];
      float val = value[m*WARPSIZE+wind];
      for (int f = 0; f < MINIBATCH; f++)
        reduce[f] += shared[f*buffsize+ind]*val;
    }
    __syncthreads();
  }
  int m = blockIdx.x*blockDim.x+threadIdx.x;
  for (int f = 0; f < MINIBATCH; f++)
    if (nextfeat[(blockIdx.y*MINIBATCH+f)*neuron+m]=__ReLU(reduce[f]+bias))
      atomicAdd(active+blockIdx.y*MINIBATCH+f,1);
    
};

void setup_gpu() {

  OR_FATAL(hipSetDevice(myid%6));

  OR_FATAL(hipStreamCreate(&copystream));
  OR_FATAL(hipStreamCreate(&kernelstream));

  buffsize = BUFFER*1024/sizeof(float)/MINIBATCH;
  numblocks = neuron/BLOCKSIZE;
  numwarp = BLOCKSIZE/WARPSIZE;

  preproc();

  double memother = 0.0;
  OR_FATAL(hipHostMalloc((void**)&globalcategories,sizeof(int)*mybatch));
  OR_FATAL(hipHostMalloc((void**)&categories,sizeof(int)*mybatch));
  OR_FATAL(hipHostMalloc((void**)&active,sizeof(int)*mybatch));
  OR_FATAL(hipMalloc((void**)&active_d,sizeof(int)*extbatch));
  OR_FATAL(hipMalloc((void**)&categories_d,sizeof(int)*extbatch));
  memother += sizeof(int)*extbatch/1.0e9;
  memother += sizeof(int)*extbatch/1.0e9;
  for (int k = 0; k < mybatch; k++) {
    active[k] = neuron;
    categories[k] = k;
    globalcategories[k] = batchdispl[myid]+k;
  }
  OR_FATAL(hipMemset(active_d,0,sizeof(int)*extbatch));
  OR_FATAL(hipMemset(categories_d,0,sizeof(int)*extbatch));
  OR_FATAL(hipMemcpy(active_d,active,sizeof(int)*mybatch,hipMemcpyHostToDevice));
  OR_FATAL(hipMemcpy(categories_d,categories,sizeof(int)*mybatch,hipMemcpyHostToDevice));

  double memweight = 0.0;
  double memdispl = 0.0;
  double memmap = 0.0;
  buffdispl_d = new int*[layer];
  mapdispl_d = new int*[layer];
  warpdispl_d = new int*[layer];
  #ifdef OUTOFCORE
  weightsizemax = 0;
  mapsizemax = 0;
  #else
  map_d = new unsigned short*[layer];
  warpindex_d = new unsigned short*[layer];
  warpvalue_d = new float*[layer];
  #endif
  for (int l = 0; l < layer; l++) {
    OR_FATAL(hipMalloc((void**)&buffdispl_d[l],sizeof(int)*(numblocks+1)));
    OR_FATAL(hipMalloc((void**)&mapdispl_d[l],sizeof(int)*(buffdispl[l][numblocks]+1)));
    OR_FATAL(hipMalloc((void**)&warpdispl_d[l],sizeof(int)*(buffdispl[l][numblocks]*numwarp+1)));
    memdispl += sizeof(int)*(numblocks+1)/1.0e9;
    memdispl += sizeof(int)*(buffdispl[l][numblocks]+1)/1.0e9;
    memdispl += sizeof(int)*(buffdispl[l][numblocks]*numwarp+1)/1.0e9;
    OR_FATAL(hipMemcpy(buffdispl_d[l],buffdispl[l],sizeof(int)*(numblocks+1),hipMemcpyHostToDevice));
    OR_FATAL(hipMemcpy(mapdispl_d[l],mapdispl[l],sizeof(int)*(buffdispl[l][numblocks]+1),hipMemcpyHostToDevice));
    OR_FATAL(hipMemcpy(warpdispl_d[l],warpdispl[l],sizeof(int)*(buffdispl[l][numblocks]*numwarp+1),hipMemcpyHostToDevice));
    #ifdef OUTOFCORE
    int mapsize = mapdispl[l][buffdispl[l][numblocks]];
    if (mapsize > mapsizemax)
      mapsizemax = mapsize;
    int weightsize = warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE;
    if (weightsize > weightsizemax)
      weightsizemax = weightsize; 
    #else
    OR_FATAL(hipMalloc((void**)&map_d[l],sizeof(unsigned short)*(mapdispl[l][buffdispl[l][numblocks]])));
    OR_FATAL(hipMalloc((void**)&warpindex_d[l],sizeof(unsigned short)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE));
    OR_FATAL(hipMalloc((void**)&warpvalue_d[l],sizeof(float)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE));
    memmap += sizeof(unsigned short)*(mapdispl[l][buffdispl[l][numblocks]])/1.0e9;
    memweight += sizeof(unsigned short)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE/1.0e9;
    memweight += sizeof(float)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE/1.0e9;
    OR_FATAL(hipMemcpy(map_d[l],map[l],sizeof(unsigned short)*(mapdispl[l][buffdispl[l][numblocks]]),hipMemcpyHostToDevice));
    OR_FATAL(hipMemcpy(warpindex_d[l],warpindex[l],sizeof(unsigned short)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE,hipMemcpyHostToDevice));
    OR_FATAL(hipMemcpy(warpvalue_d[l],warpvalue[l],sizeof(float)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE,hipMemcpyHostToDevice));
    #endif
  }
  #ifdef OUTOFCORE
  #ifdef OVERLAP
  OR_FATAL(hipMalloc((void**)&mapstream_d,sizeof(unsigned short)*mapsizemax*2));
  OR_FATAL(hipMalloc((void**)&indstream_d,sizeof(unsigned short)*weightsizemax*2));
  OR_FATAL(hipMalloc((void**)&valstream_d,sizeof(float)*weightsizemax*2));
  memmap += 2*sizeof(unsigned short)*mapsizemax/1.0e9;
  memweight += 2*sizeof(unsigned short)*weightsizemax/1.0e9;
  memweight += 2*sizeof(float)*weightsizemax/1.0e9;
  OR_FATAL(hipMemcpy(mapstream_d,map[0],sizeof(unsigned short)*mapdispl[0][buffdispl[0][numblocks]],hipMemcpyHostToDevice));
  OR_FATAL(hipMemcpy(indstream_d,warpindex[0],sizeof(unsigned short)*warpdispl[0][buffdispl[0][numblocks]*numwarp]*WARPSIZE,hipMemcpyHostToDevice));
  OR_FATAL(hipMemcpy(valstream_d,warpvalue[0],sizeof(float)*warpdispl[0][buffdispl[0][numblocks]*numwarp]*WARPSIZE,hipMemcpyHostToDevice));
  #else
  OR_FATAL(hipMalloc((void**)&mapbuff_d,sizeof(unsigned short)*mapsizemax));
  OR_FATAL(hipMalloc((void**)&indbuff_d,sizeof(unsigned short)*weightsizemax));
  OR_FATAL(hipMalloc((void**)&valbuff_d,sizeof(float)*weightsizemax));
  memmap += sizeof(unsigned short)*mapsizemax/1.0e9;
  memweight += sizeof(unsigned short)*weightsizemax/1.0e9;
  memweight += sizeof(float)*weightsizemax/1.0e9;
  #endif
  #endif

  double memfeat = 0.0;
  fprintf(stderr, "extbatch=%d, neuron=%d\n", extbatch, neuron);
  {
    const size_t bytes = sizeof(float) * size_t(extbatch) * size_t(neuron);
    fflush(stdout);
    fprintf(stderr, "hipMalloc %lu MB\n", bytes/1024/1024);
    if (hipSuccess != hipMalloc((void**)&currfeat_d,bytes)) {
      fprintf(stderr, "ERROR: need more GPU memory\n");
      exit(EXIT_FAILURE);
    }
    fprintf(stderr, "hipMalloc %lu MB\n", bytes/1024/1024);
    if (hipSuccess != hipMalloc((void**)&nextfeat_d,bytes)) {
      fprintf(stderr, "ERROR: need more GPU memory\n");
      exit(EXIT_FAILURE);
    }
    memfeat += bytes/1.0e9;
    memfeat += bytes/1.0e9;
    OR_FATAL(hipMemset(currfeat_d,0,bytes));
    OR_FATAL(hipMemset(nextfeat_d,0,bytes));
    OR_FATAL(hipMemcpy(currfeat_d,currfeat,sizeof(float)*mybatch*neuron,hipMemcpyHostToDevice));
  }

  double memothers[numproc];
  double memweights[numproc];
  double memdispls[numproc];
  double memmaps[numproc];
  double memfeats[numproc];

  memothers[0] = memother;
  memweights[0] = memweight;
  memdispls[0] = memdispl;
  memmaps[0] = memmap;
  memfeats[0] = memfeat;
}


/* 
Simultaneously launch the kernel and copy weights for the next layer.

Two streams: kernelStream and copyStream.
kernelStream contains the kernel, as well as the associated memset, and bookkeeping operations
copyStream just has the copy operations for the next layer

use copyStart / copyStop events to time the stream, and start/stop events to time the kernel

*/
void infer_gpu(int l) {

/* if OUTOFCORE and OVERLAP, point at the right part of the double-buffer to get the weights from the previous iteration
  if OUTOFCORE and !OVERLAP, copy arguments into the kernel
  otherwise, just get the right layer pointers
*/
  #ifdef OUTOFCORE
  #ifdef OVERLAP
  mapbuff_d = mapstream_d+(l%2)*mapsizemax;
  indbuff_d = indstream_d+(l%2)*weightsizemax;
  valbuff_d = valstream_d+(l%2)*weightsizemax;
  OR_FATAL(hipStreamSynchronize(copystream));
  #else
  int weightsize = warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE;
  OR_FATAL(hipMemcpyAsync(indbuff_d,warpindex[l],sizeof(unsigned short)*weightsize,hipMemcpyHostToDevice,kernelstream));
  OR_FATAL(hipMemcpyAsync(valbuff_d,warpvalue[l],sizeof(float)*weightsize,hipMemcpyHostToDevice,kernelstream));

  int mapsize = mapdispl[l][buffdispl[l][numblocks]];
  OR_FATAL(hipMemcpyAsync(mapbuff_d,map[l],sizeof(unsigned short)*mapsize,hipMemcpyHostToDevice,kernelstream));
  #endif
  #else
  mapbuff_d = map_d[l];
  indbuff_d = warpindex_d[l];
  valbuff_d = warpvalue_d[l];
  #endif

  dim3 block(BLOCKSIZE);
  dim3 grid(numblocks,(mybatch+MINIBATCH-1)/MINIBATCH);

  // initialize active features in the batch
  OR_FATAL(hipMemsetAsync(active_d,0,sizeof(int)*mybatch,kernelstream));
  dummy_kernel<<<grid,block,sizeof(float)*buffsize*MINIBATCH,kernelstream>>>(nextfeat_d,currfeat_d,buffsize,buffdispl_d[l],mapdispl_d[l],mapbuff_d,warpdispl_d[l],indbuff_d,valbuff_d,bias,neuron,categories_d,active_d);
  OR_FATAL(hipMemcpyAsync(active,active_d,sizeof(int)*mybatch,hipMemcpyDeviceToHost,kernelstream));

  #ifdef OUTOFCORE
  #ifdef OVERLAP
  if (l+1 < layer) {
    OR_FATAL(hipMemcpyAsync(mapstream_d+((l+1)%2)*mapsizemax,map[l+1],sizeof(unsigned short)*mapdispl[l+1][buffdispl[l+1][numblocks]],hipMemcpyHostToDevice,copystream));
    OR_FATAL(hipMemcpyAsync(indstream_d+((l+1)%2)*weightsizemax,warpindex[l+1],sizeof(unsigned short)*warpdispl[l+1][buffdispl[l+1][numblocks]*numwarp]*WARPSIZE,hipMemcpyHostToDevice,copystream));
    OR_FATAL(hipMemcpyAsync(valstream_d+((l+1)%2)*weightsizemax,warpvalue[l+1],sizeof(float)*warpdispl[l+1][buffdispl[l+1][numblocks]*numwarp]*WARPSIZE,hipMemcpyHostToDevice,copystream));
  }
  #else
  #endif
  #endif

  OR_FATAL(hipStreamSynchronize(kernelstream));

  int feature = 0;
  for (int k = 0; k < mybatch; k++) {
    if (active[k]) {
      globalcategories[feature] = globalcategories[k];
      categories[feature] = k;
      feature++;
    }
  }
  mybatch = feature;

  OR_FATAL(hipMemcpyAsync(categories_d,categories,sizeof(int)*feature,hipMemcpyHostToDevice,kernelstream));

  float *tempfeat_d = currfeat_d;
  currfeat_d = nextfeat_d;
  nextfeat_d = tempfeat_d;
};


void preproc() {
  buffdispl = new int*[layer];
  mapdispl = new int*[layer];
  warpdispl = new int*[layer];
  map = new unsigned short*[layer];
  warpindex = new unsigned short*[layer];
  warpvalue = new float*[layer];

  int totbuff = 0;
  int totmapnz = 0;
  int totwarpnz = 0;
  int *temptag = new int[neuron*numthreads];

  for (int l = 0; l < layer; l++) {
    int *numbuff = new int[numblocks];
    buffdispl[l] = new int[numblocks+1];
    
    #pragma omp parallel for
    for (int b = 0; b < numblocks; b++) {
      int *temp = temptag+omp_get_thread_num()*neuron;

      for (int n = 0; n < neuron; n++) {
        temp[n] = 0;
      }
        
      for (int m = b*BLOCKSIZE; m < (b+1)*BLOCKSIZE; m++) {
        for (int n = csrdispl[l][m]; n < csrdispl[l][m+1]; n++) {
          temp[csrindex[l][n]]++;
        }
      }
        
      int footprint = 0;
      for (int n = 0; n < neuron; n++) {
        if (temp[n]) {
          footprint++;
        } 
      }
      numbuff[b] = (footprint+buffsize-1)/buffsize;
    }

    buffdispl[l][0] = 0;
    for (int b = 0; b < numblocks; b++) {
      buffdispl[l][b+1] = buffdispl[l][b]+numbuff[b];
    }
      
    totbuff += buffdispl[l][numblocks];
    int *warpnz = new int[buffdispl[l][numblocks]*numwarp];
    #pragma omp parallel for
    for (int n = 0; n < buffdispl[l][numblocks]*numwarp; n++) {
      warpnz[n] = 0;
    }
      
    int *mapnz = new int[buffdispl[l][numblocks]];
    #pragma omp parallel for
    for (int n = 0; n < buffdispl[l][numblocks]; n++) {
      mapnz[n] = 0;
    }
      
    #pragma omp parallel for
    for (int b = 0; b < numblocks; b++) {
      int *temp = temptag+omp_get_thread_num()*neuron;
      for (int n = 0; n < neuron; n++) {
        temp[n] = 0;
      }
        
      for (int m = b*BLOCKSIZE; m < (b+1)*BLOCKSIZE; m++) {
        for (int n = csrdispl[l][m]; n < csrdispl[l][m+1]; n++) {
          temp[csrindex[l][n]]++;
        }
      }
        
      int footprint = 0;
      for (int n = 0; n < neuron; n++) {
        if (temp[n]) {
          int buff = footprint/buffsize;
          mapnz[buffdispl[l][b]+buff]++;
          temp[n] = buff;
          footprint++;
        }
      }
        
      for (int buff = 0; buff < numbuff[b]; buff++) {
        for (int warp = 0; warp < numwarp; warp++) {
          int tempnz[WARPSIZE] = {0};
          for (int t = 0; t < WARPSIZE; t++) {
            for (int n = csrdispl[l][b*BLOCKSIZE+warp*WARPSIZE+t]; n < csrdispl[l][b*BLOCKSIZE+warp*WARPSIZE+t+1]; n++) {
              if (temp[csrindex[l][n]]==buff) {
                tempnz[t]++;
              }
            }
          }
            
          int warpmax = 0;
          for (int t = 0; t < WARPSIZE; t++) {
            if (tempnz[t]>warpmax) {
              warpmax = tempnz[t];
            }
          }
            
          warpnz[(buffdispl[l][b]+buff)*numwarp+warp] = warpmax;
        }
      }
    }

    warpdispl[l] = new int[buffdispl[l][numblocks]*numwarp+1];
    warpdispl[l][0] = 0;
    for (int warp = 0; warp < buffdispl[l][numblocks]*numwarp; warp++) {
      warpdispl[l][warp+1] = warpdispl[l][warp]+warpnz[warp];
    }
      
    totwarpnz += warpdispl[l][buffdispl[l][numblocks]*numwarp];
    OR_FATAL(hipHostMalloc((void**)&warpindex[l],sizeof(unsigned short)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE));
    OR_FATAL(hipHostMalloc((void**)&warpvalue[l],sizeof(float)*warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE));

    #pragma omp parallel for
    for (int n = 0; n < warpdispl[l][buffdispl[l][numblocks]*numwarp]*WARPSIZE; n++) {
      warpindex[l][n] = 0;
      warpvalue[l][n] = 0.0;
    }

    mapdispl[l] = new int[buffdispl[l][numblocks]+1];
    mapdispl[l][0] = 0;

    for (int buff = 0; buff < buffdispl[l][numblocks]; buff++) {
      mapdispl[l][buff+1] = mapdispl[l][buff] + mapnz[buff];
    }
      
    totmapnz += mapdispl[l][buffdispl[l][numblocks]];
    OR_FATAL(hipHostMalloc((void**)&map[l],sizeof(unsigned short)*mapdispl[l][buffdispl[l][numblocks]]));

    #pragma omp parallel for
    for (int n = 0; n < buffdispl[l][numblocks]; n++) {
      mapnz[n] = 0;
    }
      
    #pragma omp parallel for
    for (int b = 0; b < numblocks; b++) {
      int *temp = temptag+omp_get_thread_num()*neuron;
      for (int n = 0; n < neuron; n++) {
        temp[n] = 0;
      }
        
      for (int m = b*BLOCKSIZE; m < (b+1)*BLOCKSIZE; m++) {
        for (int n = csrdispl[l][m]; n < csrdispl[l][m+1]; n++) {
          temp[csrindex[l][n]]++;
        } 
      }
        
      int footprint = 0;
      for (int n = 0; n < neuron; n++) {
        if (temp[n]) {
          int buff = footprint/buffsize;
          map[l][mapdispl[l][buffdispl[l][b]+buff]+mapnz[buffdispl[l][b]+buff]] = n;
          mapnz[buffdispl[l][b]+buff]++;
          temp[n] = footprint;
          footprint++;
        }
      }
        
      for (int buff = 0; buff < numbuff[b]; buff++) {
        for (int warp = 0; warp < numwarp; warp++) {
          int tempnz[WARPSIZE] = {0};
          for (int t = 0; t < WARPSIZE; t++) {
            for (int n = csrdispl[l][b*BLOCKSIZE+warp*WARPSIZE+t]; n < csrdispl[l][b*BLOCKSIZE+warp*WARPSIZE+t+1]; n++) {
              if (temp[csrindex[l][n]]/buffsize==buff) {
                 int ind = (warpdispl[l][(buffdispl[l][b]+buff)*numwarp+warp]+tempnz[t])*WARPSIZE+t;
                 warpindex[l][ind] = temp[csrindex[l][n]]%buffsize;
                 warpvalue[l][ind] = csrvalue[l][n];
                 tempnz[t]++;
              }
            }
          }
        }
      }   
    }

    delete[] numbuff;
    delete[] mapnz;
    delete[] warpnz;
    delete[] csrdispl[l];
    delete[] csrindex[l];
    delete[] csrvalue[l];
  }

  delete[] temptag;
  delete[] csrdispl;
  delete[] csrindex;
  delete[] csrvalue;
};
